#include "hip/hip_runtime.h"
#include <stddef.h>
#include <stdint.h>

constexpr const int num_buckets = 16;

constexpr const int top_step_x = 32;
cosntexpr const int top_step_y = 4;

__global__ void tree_top_accumulator(const int64_t nrows, const int64_t nfolds,
                                     const int64_t npicks, const float *const g,
                                     const int64_t *const picked_features,
                                     const uint8_t *const x,
                                     const size_t x_pitch,
                                     float *const output) {
  int j = blockIdx.x;
  int f = blockIdx.y;

  int group_idx = threadIdx.x + threadIdx.y * top_step_x;

  const int nrows_over_2 = nrows / 2;

  const uint8_t *this_x = x + x_pitch * picked_features[f];
  const float *g_for_j = g + j * nrows;

  __shared__ float all_temp_output[num_buckets * top_step_x * top_step_y];
  float *this_temp_out = all_temp_output + group_idx;

  for (int i = 0; i < num_buckets; i++) {
    this_temp_out[i * top_step_x * top_step_y] = 0.0f;
  }

  for (int idx = group_idx * 4; idx < nrows_over_2;
       idx += top_step_x * top_step_y * 4) {
    int this_int = *reinterpret_cast<const int *>(this_x + idx);
    float4 g0 = *reinterpret_cast<const float4 *>(g_for_j + idx * 2);
    float4 g1 = *reinterpret_cast<const float4 *>(g_for_j + idx * 2 + 4);

    int val_0_x = (this_int >> 0x00) & 0x0f;
    int val_0_y = (this_int >> 0x04) & 0x0f;
    int val_0_z = (this_int >> 0x08) & 0x0f;
    int val_0_w = (this_int >> 0x0c) & 0x0f;
    int val_1_x = (this_int >> 0x10) & 0x0f;
    int val_1_y = (this_int >> 0x14) & 0x0f;
    int val_1_z = (this_int >> 0x18) & 0x0f;
    int val_1_w = (this_int >> 0x1c) & 0x0f;

    this_temp_out[val_0_x * top_step_x * top_step_y] += g0.x;
    this_temp_out[val_0_y * top_step_x * top_step_y] += g0.y;
    this_temp_out[val_0_z * top_step_x * top_step_y] += g0.z;
    this_temp_out[val_0_w * top_step_x * top_step_y] += g0.w;
    this_temp_out[val_1_x * top_step_x * top_step_y] += g1.x;
    this_temp_out[val_1_y * top_step_x * top_step_y] += g1.y;
    this_temp_out[val_1_z * top_step_x * top_step_y] += g1.z;
    this_temp_out[val_1_w * top_step_x * top_step_y] += g1.w;
  }

  float *output_for_j = output + f * num_buckets + j * npick * num_buckets;

  for (int i = 0; i < num_buckets; i++) {
    atomicAdd(output_for_j + i, this_temp_out[i * top_step_x * top_step_y]);
  }
}
